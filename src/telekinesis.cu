#include "hip/hip_runtime.h"
#include "cudarray.cuh"
#include "telekinesis.cuh"
#include <cstdint>
#include <sys/types.h>

template <typename T>
__global__ void kernel(T *vec, T scalar, int num_elements) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements) {
    vec[idx] = vec[idx] * scalar;
  }
}

template <typename T> void run_kernel(T *vec, T scalar, int num_elements) {
  dim3 dimBlock(1024, 1, 1);
  dim3 dimGrid(ceil((T)num_elements / dimBlock.x));

  kernel<T><<<dimGrid, dimBlock>>>(vec, scalar, num_elements);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::stringstream strstr;
    strstr << "run_kernel launch failed" << std::endl;
    strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
    strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
    strstr << hipGetErrorString(error);
    throw strstr.str();
  }
}

template <typename T>
uintptr_t array_create(pybind11::array_t<T> vec) {
  pybind11::buffer_info ha = vec.request();

  if (ha.ndim != 1) {
    std::stringstream strstr;
    strstr << "ha.ndim != 1" << std::endl;
    strstr << "ha.ndim: " << ha.ndim << std::endl;
    throw std::runtime_error(strstr.str());
  }

  int size = ha.shape[0];
  int size_bytes = size * sizeof(T);
  T *gpu_ptr;
  hipError_t error = hipMalloc(&gpu_ptr, size_bytes);

  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  T *ptr = reinterpret_cast<T *>(ha.ptr);
  error = hipMemcpy(gpu_ptr, ptr, size_bytes, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  hipError_t cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(cuda_status));
  }

  return reinterpret_cast<uintptr_t>(gpu_ptr);
}

template <typename T>
void array_map(uintptr_t gpu_ptr_python, T scalar, int size) {
  T *gpu_ptr = reinterpret_cast<T *>(gpu_ptr_python);
  
  run_kernel<T>(gpu_ptr, scalar, size);
}

template <typename T>
void array_remove(uintptr_t gpu_ptr_python, pybind11::array_t<T> vec) {
  pybind11::buffer_info ha = vec.request();

  if (ha.ndim != 1) {
    std::stringstream strstr;
    strstr << "ha.ndim != 1" << std::endl;
    strstr << "ha.ndim: " << ha.ndim << std::endl;
    throw std::runtime_error(strstr.str());
  }

  int size = ha.shape[0];
  int size_bytes = size * sizeof(T);

  T *gpu_ptr = reinterpret_cast<T *>(gpu_ptr_python);
  T *ptr = reinterpret_cast<T *>(ha.ptr);

  hipError_t cuda_status = hipDeviceSynchronize();
  if (cuda_status != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(cuda_status));
  }

  hipError_t error = hipMemcpy(ptr, gpu_ptr, size_bytes, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  error = hipFree(gpu_ptr);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

PYBIND11_MODULE(telekinesis, m) {
  m.def("array_create", &array_create<double>);
  m.def("array_map", &array_map<double>);
  m.def("array_remove", &array_remove<double>);
}
